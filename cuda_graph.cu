bool graphCreated=false;
hipGraph_t graph;
hipGraphExec_t instance;
for(int istep=0; istep<NSTEP; istep++){
  if(!graphCreated){
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
      shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
    }
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
    graphCreated=true;
  }
  hipGraphLaunch(instance, stream);
  hipStreamSynchronize(stream);
}
