#include "hip/hip_runtime.h"
__global__ void Kernel(int count_a, int count_b)
{
    extern __shared__ int *shared;
    int *a = &shared[0]; //a is manually set at the beginning of shared
    int *b = &shared[count_a]; //b is manually set at the end of a
}

sharedMemory = count_a*size(int) + size_b*size(int);
Kernel <<<numBlocks, threadsPerBlock, sharedMemory>>> (count_a, count_b);
